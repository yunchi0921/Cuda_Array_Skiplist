#include"Skiplist.h"

Node* test_Init(Node *sl,Node *n_arr,int N){
	Init <<<1,20>>> (sl,n_arr,N);
	return sl;
}
Node* test_Connect(Node*sl,int N){
	Connect<<<1,20>>>(sl,N);
	return sl;
}
int main(){
	int N=5;
	Node* sl=(Node*)malloc(N*MAX_LEVEL*sizeof(Node));
	Node* d_sl;
	Node* n_arr=(Node*)malloc(N*sizeof(Node));
	Node* d_n_arr;
	//hipEvent_t start,stop;
	//float elapsedTime = 0.0;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);

	for(int i=0 ; i<MAX_LEVEL*N ;i++){
				sl[i].key=0;
				sl[i].level=0;
				sl[i].nextIdx=0;
			}

	srand(time(NULL));
	for(int i=0;i<N;i++){
		n_arr[i].key=i;
		n_arr[i].level=rand()%MAX_LEVEL+1;
	}


	hipMalloc(&d_sl,N*MAX_LEVEL*sizeof(Node));
	hipMalloc(&d_n_arr,N*sizeof(Node));
	//counting time start
	//hipEventRecord(start,0);
	hipMemcpy(d_sl,sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyHostToDevice);
	hipMemcpy(d_n_arr,n_arr,N*sizeof(Node),hipMemcpyHostToDevice);
	test_Init(d_sl,d_n_arr,N);
	test_Connect(d_sl,N);
	hipMemcpy(sl,d_sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyDeviceToHost);
	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsedTime,start,stop);

	//printf("elapsedTime : %f\n",elapsedTime);

	for(int i=0 ; i<MAX_LEVEL*N ;i++){

		printf("%d ",sl[i].key);
		if(i%N==N-1)
		printf("\n");
	}
	for(int i=0 ;i<MAX_LEVEL*N;i++){
		printf("%d ",sl[i].nextIdx%N);
		if(i%N==N-1)
			printf("\n");
	}
	free(sl);
	free(n_arr);
	hipFree(d_sl);
	hipFree(d_n_arr);
}
