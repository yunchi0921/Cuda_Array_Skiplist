#include "hip/hip_runtime.h"
#include"Skiplist.h"
void __global__ add(Node **sl,Node *n, int *latest){
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int first_read,second_read;
	int new_latest=*latest+1;
	do{
		first_read=*latest;
		//Call Insert
		Insert(sl,n[x],first_read);
		//Assert latest haven't been changed
		second_read=atomicCAS((unsigned long long int*)latest,
				*(unsigned long long int *)&first_read,
				*(unsigned long long int *)&new_latest);
	}while(first_read!=second_read); //If latest was been changed , rework.
}
