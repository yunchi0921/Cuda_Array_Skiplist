#include "Skiplist.h"
#include <thrust/sort.h>
#include <assert.h>
void test_Init(Node *sl, int N, int BLOCKSIZE, int GRIDSIZE) {
  Init<<<N / BLOCKSIZE, BLOCKSIZE, BLOCKSIZE * sizeof(Node)>>>(sl, N);
}
void test_Connect(Node *sl, int N, int BLOCKSIZE, int GRIDSIZE) {
  Connect<<<GRIDSIZE, BLOCKSIZE, BLOCKSIZE * sizeof(Node)>>>(sl, N);
}
void shuffle(int *a, int n) {
  int i, j, tmp, T = 1000;
  while (T--) {
    i = rand() % n;
    j = rand() % n;
    tmp = a[i];
    a[i] = a[j];
    a[j] = tmp;
  }
}
struct timespec diff(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec - start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec - start.tv_sec - 1;
    temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  }
  return temp;
}

struct timespec add(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec + start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec + start.tv_sec + 1;
    temp.tv_nsec = end.tv_nsec + start.tv_nsec - 1000000000;
  } else {
    temp.tv_sec = end.tv_sec + start.tv_sec;
    temp.tv_nsec = end.tv_nsec + start.tv_nsec;
  }
  return temp;
}

int main(int argc, char *argv[]) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);
  if (argc < 3) {
    printf("error:Need more argument\n");
    return 0;
  }
  int gridsize = atoi(argv[1]);
  int blocksize = atoi(argv[2]);
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  int N = gridsize * blocksize;
  Node *sl;
  Node *d_sl;
  struct timespec time1, time2, temp;
  double time_used, sum = 0,sum_sort=0;
  int loop;
  int *input;
  thrust::host_vector<int> h_s(N);
  thrust::device_vector<int> d_s(N);
  for (loop = 1; loop <=1; loop++) {
    input = (int *)malloc(N * sizeof(int));
    // initializtion
    sl = (Node *)malloc(N * MAX_LEVEL * sizeof(Node));
    err = hipMalloc(&d_sl, N * MAX_LEVEL * sizeof(Node));
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to allocate device skiplist  (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    for (int i = 0; i < N; i++) {
      input[i] = i;
    }
    shuffle(input, N); // random the order
    thrust::host_vector<int> h_s(N);
    //give number to host_vector
    for(int i=0;i<N;i++){
    	h_s[i]=input[i];
    }
    d_s=h_s;

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
    thrust::sort(d_s.begin(), d_s.end()); // sorting
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time2);
    temp = diff(time1, time2);
    time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
    sum_sort+=time_used;
    //copy device to host
    h_s=d_s;

    //check 
    for(int i=0;i<N;i++){
    	assert(h_s[i]==i);
    }
    srand(time(NULL)); // to rand the level
    for (int i = 0; i < N * MAX_LEVEL; i++) {
      if (i < N) {
        sl[i].key = h_s[i];
        if (i % blocksize == blocksize - 1 || i % blocksize == 0) {
          sl[i].level = MAX_LEVEL;
        } else {
          sl[i].level = rand() % MAX_LEVEL + 1;
        }
      } else {
        sl[i].key = -1;
      }
      if (i % blocksize == blocksize - 1)
        sl[i].nextIdx = i + 1;
      else
        sl[i].nextIdx = -1;
    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1); // timespec start
    err = hipMemcpy(d_sl, sl, N * MAX_LEVEL * sizeof(Node),
                     hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy device skiplist from host to device (error code "
              "%s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    test_Init(d_sl, N, blocksize, gridsize);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch Init kernel (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    test_Connect(d_sl, N, blocksize, gridsize);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch connect kernel (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    err = hipMemcpy(sl, d_sl, N * MAX_LEVEL * sizeof(Node),
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy device skiplist from device to host (error code "
              "%s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2); // timespec stop
    temp = diff(time1, time2);
    time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
    sum += time_used;
    printf("Skiplist node value:\n");
    for (int i = 0; i < MAX_LEVEL * N; i++) {
            printf("%2d ", sl[i].key);
            if (i % N == N - 1)
                    printf("\n");
    }
    printf("Skiplist nextIdx:\n");
    for (int i = 0; i < MAX_LEVEL * N; i++) {
            printf("%2d ", sl[i].nextIdx % N);
            if (i % N == N - 1)
                    printf("\n");
    }

    free(input);
    free(sl);
    err = hipFree(d_sl);
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to free device skiplist (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    
  }
  printf("Sorting time= %f\n", sum_sort/loop);
  printf("%d\t%f\n", N, sum / loop);
}
