#include"Skiplist.h"
#include <thrust/sort.h>
#define BLOCKSIZE 1024
#define GRIDSIZE 16



void test_Init(Node *sl,int N) {
	Init<<<N/BLOCKSIZE,BLOCKSIZE, BLOCKSIZE * sizeof(Node)>>>(
			sl,N);

}
void test_Connect(Node*sl, int N) {
	Connect<<<GRIDSIZE, BLOCKSIZE, BLOCKSIZE * sizeof(Node)>>>(sl, N);
}
void shuffle(int *a,int n){
	int i,j,tmp,T=1000;
	while(T--)
	{
		i=rand()%n;
		j=rand()%n;
		tmp=a[i];
		a[i]=a[j];
		a[j]=tmp;
	}
}
struct timespec diff(timespec start, timespec end) {
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0) {
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	return temp;
}

struct timespec add(timespec start, timespec end) {
	struct timespec temp;
	if ((end.tv_nsec + start.tv_nsec) < 0) {
		temp.tv_sec = end.tv_sec + start.tv_sec + 1;
		temp.tv_nsec = end.tv_nsec + start.tv_nsec - 1000000000;
	} else {
		temp.tv_sec = end.tv_sec + start.tv_sec;
		temp.tv_nsec = end.tv_nsec + start.tv_nsec;
	}
	return temp;
}

int main() {
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
	int N=BLOCKSIZE*GRIDSIZE;
	Node* sl;
	Node* d_sl;
	struct timespec time1, time2;
	struct timespec sum;
	int loop = 1;
	int* input=(int*)malloc(N*sizeof(int));
	//initializtion
	sl = (Node*) malloc(N * MAX_LEVEL * sizeof(Node));
	err = hipMalloc(&d_sl, N * MAX_LEVEL * sizeof(Node));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate device skiplist  (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	for(int i=0;i<N;i++){
		input[i]=i;
	}
	shuffle(input,N);//random the order
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
	thrust::sort(input,input+N);//sorting
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);
	struct timespec temp=diff(time1,time2);
	double time_used=1000*(temp.tv_sec+(double)temp.tv_nsec/1000000000.0);
	printf("Sorting time= %f\n",time_used);
	srand(time(NULL)); // to rand the level
	for (int i = 0; i < N * MAX_LEVEL; i++) {
		if (i < N) {
			sl[i].key = input[i];
			if (i % BLOCKSIZE == BLOCKSIZE-1 || i % BLOCKSIZE == 0) {
				sl[i].level = MAX_LEVEL;
			} else {
				sl[i].level = rand() % MAX_LEVEL + 1;
			}
		} else {
			sl[i].key = -1;
		}
		if (i % BLOCKSIZE == BLOCKSIZE - 1)
			sl[i].nextIdx = i + 1;
		else
			sl[i].nextIdx = -1;
	}
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1); //timespec start
	err = hipMemcpy(d_sl, sl, N * MAX_LEVEL * sizeof(Node),
			hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy device skiplist from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	test_Init(d_sl,N);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch Init kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	test_Connect(d_sl, N);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch connect kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(sl, d_sl, N * MAX_LEVEL * sizeof(Node),
			hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy device skiplist from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2); //timespec stop
	sum = add(time1, time2);
	/*printf("Skiplist node value:\n");
	for (int i = 0; i < MAX_LEVEL * N; i++) {
		printf("%2d ", sl[i].key);
		if (i % N == N - 1)
			printf("\n");
	}
	printf("Skiplist nextIdx:\n");
	for (int i = 0; i < MAX_LEVEL * N; i++) {
		printf("%2d ", sl[i].nextIdx % N);
		if (i % N == N - 1)
			printf("\n");
	}*/

	printf("%d\t%ld\n",N,(sum.tv_sec*1000000000+sum.tv_nsec)/loop);
	free(sl);
	err = hipFree(d_sl);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device skiplist (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

