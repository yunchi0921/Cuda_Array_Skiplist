#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Skiplist.cu
 Author      : Yunchi
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include"Skiplist.h"

/*Parallelize to Initial all elements of array*/
void __global__ Init(Node *sl, int N) {
	extern __shared__ Node s[];
	int g = blockIdx.x * blockDim.x + threadIdx.x;
	int x = threadIdx.x;
	Node *shared_level_zero = s;
	shared_level_zero[x] = sl[g];
	int level = shared_level_zero[x].level;
	for(int i=0;i<level;i++){
			sl[g+N*i].key = shared_level_zero[x].key;
	}
}
void __global__ Connect(Node*sl, int N) {
	int g=blockIdx.x*blockDim.x+threadIdx.x;
	int x=threadIdx.x;
	extern __shared__ Node s[];
	Node* shared_level_zero=s;
	shared_level_zero[x]=sl[g];
	int level=shared_level_zero[x].level;
	if(x!=blockDim.x-1){
		int step=1;
		for(int i=0;i<level;i++){
			while(shared_level_zero[x+step].level<=i && x+step<blockDim.x) step++;
			sl[g+N*i].nextIdx=g+step;
		}
	}
}

