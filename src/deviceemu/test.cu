#include"Skiplist.h"
#include<iostream>
using namespace std;
#define BLOCKSIZE 512
#define GRIDSIZE 65535
Node* test_Init(Node *sl,Node *n_arr,int N){
	Init <<<GRIDSIZE,BLOCKSIZE>>> (sl,n_arr,N);
	return sl;
}
Node* test_Connect(Node*sl,int N){
	Connect<<<GRIDSIZE,BLOCKSIZE>>>(sl,N);
	return sl;
}
struct timespec diff(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec - start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec - start.tv_sec - 1;
    temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  }
  return temp;
}

int main(){
	int N=BLOCKSIZE*GRIDSIZE/MAX_LEVEL;
	Node* sl=(Node*)malloc(N*MAX_LEVEL*sizeof(Node));
	Node* d_sl;
	Node* n_arr=(Node*)malloc(N*sizeof(Node));
	Node* d_n_arr;
	struct timespec time1,time2;

	for(int i=0 ; i<MAX_LEVEL*N ;i++){
				sl[i].key=-1;
				sl[i].level=0;
				sl[i].nextIdx=-1;
			}

	srand(time(NULL));
	for(int i=0;i<N;i++){
		n_arr[i].key=i;
		n_arr[i].level=rand()%MAX_LEVEL+1;
	}


	hipMalloc(&d_sl,N*MAX_LEVEL*sizeof(Node));
	hipMalloc(&d_n_arr,N*sizeof(Node));

	clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time1);//timespec start
	hipMemcpy(d_sl,sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyHostToDevice);
	hipMemcpy(d_n_arr,n_arr,N*sizeof(Node),hipMemcpyHostToDevice);
	test_Init(d_sl,d_n_arr,N);
	test_Connect(d_sl,N);
	hipMemcpy(sl,d_sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyDeviceToHost);
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time2);//timespec stop

	printf("elapsedTime using timespec: %lld.%ld\n",(long long)diff(time1,time2).tv_sec,diff(time1,time2).tv_nsec);
/*	printf("Skiplist node value:\n");
	for(int i=0 ; i<MAX_LEVEL*N ;i++){
		printf("%2d ",sl[i].key);
		if(i%N==N-1)
		printf("\n");
	}
	printf("Skiplist nextIdx:\n");
	for(int i=0 ;i<MAX_LEVEL*N;i++){
		printf("%2d ",sl[i].nextIdx%N);
		if(i%N==N-1)
			printf("\n");
	}*/
	free(sl);
	free(n_arr);
	hipFree(d_sl);
	hipFree(d_n_arr);
}
