#include"Skiplist.h"
#include<iostream>
using namespace std;
#define BLOCKSIZE 512
#define GRIDSIZE 8192
Node* test_Init(Node *sl,Node *n_arr,int N){
	Init <<<GRIDSIZE,BLOCKSIZE>>> (sl,n_arr,N);
	return sl;
}
Node* test_Connect(Node*sl,int N){
	Connect<<<GRIDSIZE,BLOCKSIZE>>>(sl,N);
	return sl;
}
struct timespec diff(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec - start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec - start.tv_sec - 1;
    temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  }
  return temp;
}

struct timespec add(timespec start, timespec end) {
	struct timespec temp;
	if((end.tv_nsec+start.tv_nsec) < 0) {
		temp.tv_sec = end.tv_sec+start.tv_sec + 1;
		temp.tv_nsec = end.tv_nsec + start.tv_nsec - 1000000000;
	}else{
		temp.tv_sec = end.tv_sec + start.tv_sec;
		temp.tv_nsec = end.tv_nsec + start.tv_nsec;
	}
	return temp;
}

int main(){
	int N;
	Node* sl;
	Node* d_sl;
	Node* n_arr;
	Node* d_n_arr;
	struct timespec time1,time2;
	int loop;
	struct timespec sum;
	for(int size=1;size<=GRIDSIZE;size<<=1){
		//initializtion
		N=BLOCKSIZE*size/MAX_LEVEL;
		sl=(Node*)malloc(N*MAX_LEVEL*sizeof(Node));
		n_arr=(Node*)malloc(N*sizeof(Node));
		hipMalloc(&d_sl,N*MAX_LEVEL*sizeof(Node));
		hipMalloc(&d_n_arr,N*sizeof(Node));
		for(int i=0 ; i<MAX_LEVEL*N ;i++){
					sl[i].key=-1;
					sl[i].level=0;
					sl[i].nextIdx=-1;
				}

		srand(time(NULL));
		for(int i=0;i<N;i++){
			n_arr[i].key=i;
			n_arr[i].level=rand()%MAX_LEVEL+1;
		}

		//test loop
		for(loop=1;loop<=1;loop++){
			clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time1);//timespec start
			hipMemcpy(d_sl,sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyHostToDevice);
			hipMemcpy(d_n_arr,n_arr,N*sizeof(Node),hipMemcpyHostToDevice);
			test_Init(d_sl,d_n_arr,N);
			test_Connect(d_sl,N);
			hipMemcpy(sl,d_sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyDeviceToHost);
			clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time2);//timespec stop
			sum = add(time1,time2);
			/*printf("Skiplist node value:\n");
			for(int i=0 ; i<MAX_LEVEL*N ;i++){
				printf("%2d ",sl[i].key);
				if(i%N==N-1)
				printf("\n");
			}
			printf("Skiplist nextIdx:\n");
			for(int i=0 ;i<MAX_LEVEL*N;i++){
				printf("%2d ",sl[i].nextIdx%N);
				if(i%N==N-1)
					printf("\n");
			}*/
		}
		printf("%d\t0.%.9ld\n",size,(sum.tv_sec*1000000000+sum.tv_nsec)/loop);
		free(sl);
		free(n_arr);
		hipFree(d_sl);
		hipFree(d_n_arr);
	}
}
