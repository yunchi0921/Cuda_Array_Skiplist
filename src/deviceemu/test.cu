#include"Skiplist.h"

Node* test_Init(Node *sl,Node *n_arr,int N){
	Init <<<1,256>>> (sl,n_arr,N);
	return sl;
}
Node* test_Connect(Node*sl,int N){
	Connect<<<1,256>>>(sl,N);
	return sl;
}
int main(){
	int N=10;
	Node* sl=(Node*)malloc(N*MAX_LEVEL*sizeof(Node));
	Node* d_sl;
	Node* n_arr=(Node*)malloc(N*sizeof(Node));
	Node* d_n_arr;

		for(int i=0 ; i<MAX_LEVEL*N ;i++){
				sl[i].key=0;
				sl[i].level=0;
				sl[i].nextIdx=0;
				sl[i].selfIdx=0;
			}

	srand(time(NULL));
	for(int i=0;i<10;i++){
		n_arr[i].key=i;
		n_arr[i].level=rand()%8+1;
	}


	hipMalloc(&d_sl,N*MAX_LEVEL*sizeof(Node));
	hipMalloc(&d_n_arr,N*sizeof(Node));
	hipMemcpy(d_sl,sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyHostToDevice);
	hipMemcpy(d_n_arr,n_arr,N*sizeof(Node),hipMemcpyHostToDevice);
	test_Init(d_sl,d_n_arr,N);
	test_Connect(d_sl,N);
	hipMemcpy(sl,d_sl,N*MAX_LEVEL*sizeof(Node),hipMemcpyDeviceToHost);


	for(int i=0 ; i<MAX_LEVEL*N ;i++){

		printf("%d ",sl[i].key);
		if(i%N==N-1)
		printf("\n");
	}
	for(int i=0 ;i<MAX_LEVEL*N;i++){
		printf("%d ",sl[i].nextIdx%N);
		if(i%N==N-1)
			printf("\n");
	}
}
