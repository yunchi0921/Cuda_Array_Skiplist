#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Skiplist.cu
 Author      : Yunchi
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include"Skiplist.h"



/*Parallelize to Initial all elements of array*/
void __global__ Init(Node *sl,Node *n_arr,int N){

	int x =blockIdx.x*blockDim.x+threadIdx.x;

	int level=n_arr[x%N].level;
	if((x/N)<level)
		sl[x].key=n_arr[x%N].key;
}
void __global__ Connect(Node*sl,int N){
	int x=blockIdx.x*blockDim.x+threadIdx.x;
	 if(sl[x].key!=-1 && x%N!=N-1){
		 int i=0;
		 do{
			 ++i;
		 }while(sl[x+i].key==0&&(x+i)%N!=0);
		 if((x+i)%N!=0)
			 sl[x].nextIdx=x+i;
		 else
			 sl[x].nextIdx=-1;
	 }
	 else
		 sl[x].nextIdx=-1;
}








