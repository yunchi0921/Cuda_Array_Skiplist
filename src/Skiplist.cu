#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Skiplist.cu
 Author      : Yunchi
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include"Skiplist.h"

#define MAX_LEVEL 12

void __device__ NodeSearch(Node **n,int key,int * update){

	//setting head
	Node cur=n[0][MAX_LEVEL];
	Node next_node;
	int level;

	for(level=MAX_LEVEL-1;level>=0;level--){
		next_node=n[cur.nextIdx][level];
		while(next_node.key!=INT_MAX&&next_node.key<key){
			cur=next_node;
			next_node=n[cur.nextIdx][level];
		}
		update[level]=cur.selfIdx;
	}
}

void __device__ Insert(Node **sl,Node n,int latest){
	int level=n.level;
	Node dest;
	int idx_first_read,idx_second_read;
	int update[MAX_LEVEL];
		do{
			NodeSearch(sl,n.key,update);
			for(int i=0;i<level;i++){
				dest=sl[update[i]][i];
				idx_first_read=dest.nextIdx;

				//Allow each node thread to set the forward facing index
				n.nextIdx=idx_first_read;

				idx_second_read
				=(int)atomicCAS((unsigned long long int*)&(dest.nextIdx),
						*(unsigned long long int *)&idx_first_read,
						*(unsigned long long int *)&latest);
			}
		}while(idx_first_read!=idx_second_read);

}

Node NodeCreate(int key,int level){
	Node n;
	n.key=key;
	n.level=level;
	return n;
}

Node __device__ IndexSetting(Node &n,int nextIdx,int latest ){
	n.nextIdx=nextIdx;
	n.selfIdx=latest;
	return n;
}






