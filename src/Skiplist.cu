#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Skiplist.cu
 Author      : Yunchi
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include"Skiplist.h"



/*Parallelize to Initial all elements of array*/
void __global__ Init(Node *sl,Node *n_arr,int N){

	int x =threadIdx.x+blockIdx.x*blockDim.x;
	int level=n_arr[x%N].level;
	if((x/N)<level)
		sl[x].key=n_arr[x%N].key;
}








