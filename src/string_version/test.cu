#include "Skiplist.h"
#include <thrust/sort.h>
#include <assert.h>
void test_Init(Node *sl, int N, int BLOCKSIZE, int GRIDSIZE) {
  Init<<<N / BLOCKSIZE, BLOCKSIZE, BLOCKSIZE * sizeof(Node)>>>(sl, N);
}
void test_Connect(Node *sl, int N, int BLOCKSIZE, int GRIDSIZE) {
  Connect<<<GRIDSIZE, BLOCKSIZE, BLOCKSIZE * sizeof(Node)>>>(sl, N);
}
void shuffle(vector<string> a,vector<string> b, int n) {
  int i, j, T = 1000;
  string tmp;
  while (T--) {
    i = rand() % n;
    j = rand() % n;
    tmp = a[i];
    a[i] = a[j];
    a[j] = tmp;
    tmp = b[i];
    b[i] = b[j];
    b[j] = tmp;
  }
}
struct timespec diff(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec - start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec - start.tv_sec - 1;
    temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  }
  return temp;
}

struct timespec add(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec + start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec + start.tv_sec + 1;
    temp.tv_nsec = end.tv_nsec + start.tv_nsec - 1000000000;
  } else {
    temp.tv_sec = end.tv_sec + start.tv_sec;
    temp.tv_nsec = end.tv_nsec + start.tv_nsec;
  }
  return temp;
}

int main(int argc, char *argv[]) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);
  if (argc < 3) {
    printf("error:Need more argument\n");
    return 0;
  }
  int gridsize = atoi(argv[1]);
  int blocksize = atoi(argv[2]);
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  int N = gridsize * blocksize;
  Node *sl;
  Node *d_sl;
  struct timespec time1, time2, temp;
  double time_used, sum = 0,sum_sort=0;
  int loop;
  vector<string> key,value;
  thrust::host_vector<string> h_key(N);
  thrust::host_vector<string> h_value(N);
  thrust::device_vector<string> d_key(N);
  thrust::device_vector<string> d_value(N);
  for (loop = 1; loop <=1; loop++) {
    //input = (string *)malloc(N * sizeof(string));
    // initializtion
    sl = (Node *)malloc(N * MAX_LEVEL * sizeof(Node));
    err = hipMalloc(&d_sl, N * MAX_LEVEL * sizeof(Node));
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to allocate device skiplist  (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    for (int i = 0; i < N; i++) {
      key[i] = to_string(i);
      value[i] = to_string(i);
    }
    shuffle(key,value, N); // random the order
    for(int i=0;i<N;i++){
    	h_key[i]=key[i];
	h_value[i]=value[i];
    }

    d_key=h_key;
    d_value=h_value;

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
    thrust::sort_by_key(d_key.begin(), d_key.end(),d_value.begin()); // sorting
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time2);
    temp = diff(time1, time2);
    time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
    sum_sort+=time_used;
    //copy device to host
    h_key=d_key;
    h_value=d_value;
    //check 
    for(int i=0;i<N;i++){
    	assert(h_key[i]==to_string(i));
	assert(h_value[i]==to_string(i));
    }
    srand(time(NULL)); // to rand the level
    for (int i = 0; i < N * MAX_LEVEL; i++) {
      if (i < N) {
        sl[i].key = h_key[i];
	sl[i].value=h_value[i];
        if (i % blocksize == blocksize - 1 || i % blocksize == 0) {
          sl[i].level = MAX_LEVEL;
        } else {
          sl[i].level = rand() % MAX_LEVEL + 1;
        }
      } else {
        sl[i].key = -1;
	sl[i].value = -1;
      }
      if (i % blocksize == blocksize - 1)
        sl[i].nextIdx = i + 1;
      else
        sl[i].nextIdx = -1;
    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1); // timespec start
    err = hipMemcpy(d_sl, sl, N * MAX_LEVEL * sizeof(Node),
                     hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy device skiplist from host to device (error code "
              "%s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    test_Init(d_sl, N, blocksize, gridsize);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch Init kernel (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    test_Connect(d_sl, N, blocksize, gridsize);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch connect kernel (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    err = hipMemcpy(sl, d_sl, N * MAX_LEVEL * sizeof(Node),
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr,
              "Failed to copy device skiplist from device to host (error code "
              "%s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2); // timespec stop
    temp = diff(time1, time2);
    time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
    sum += time_used;
    /*printf("Skiplist node value:\n");
    for (int i = 0; i < MAX_LEVEL * N; i++) {
            printf("%2d ", sl[i].key);
            if (i % N == N - 1)
                    printf("\n");
    }
    printf("Skiplist nextIdx:\n");
    for (int i = 0; i < MAX_LEVEL * N; i++) {
            printf("%2d ", sl[i].nextIdx % N);
            if (i % N == N - 1)
                    printf("\n");
    }*/

   // free(key);
    //free(value);
    free(sl);
    err = hipFree(d_sl);
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to free device skiplist (error code %s)!\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    
  }
  printf("Sorting time= %f\n", sum_sort/loop);
  printf("%d\t%f\n", N, sum / loop);
}
