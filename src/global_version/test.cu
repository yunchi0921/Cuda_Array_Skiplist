#include "Skiplist.h"
#include <thrust/sort.h>
#include <assert.h>
Node *test_Init(Node *sl, Node *n_arr, int N, int gridsize, int blocksize) {
  Init<<<gridsize, blocksize>>>(sl, n_arr, N);
  return sl;
}
Node *test_Connect(Node *sl, int N, int girdsize, int blocksize) {
  Connect<<<girdsize, blocksize>>>(sl, N);
  return sl;
}
void shuffle(int *a, int n) {
  int i, j, tmp, T = 1000;
  while (T--) {
    i = rand() % n;
    j = rand() % n;
    tmp = a[i];
    a[i] = a[j];
    a[j] = tmp;
  }
}
struct timespec diff(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec - start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec - start.tv_sec - 1;
    temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  }
  return temp;
}

struct timespec add(timespec start, timespec end) {
  struct timespec temp;
  if ((end.tv_nsec + start.tv_nsec) < 0) {
    temp.tv_sec = end.tv_sec + start.tv_sec + 1;
    temp.tv_nsec = end.tv_nsec + start.tv_nsec - 1000000000;
  } else {
    temp.tv_sec = end.tv_sec + start.tv_sec;
    temp.tv_nsec = end.tv_nsec + start.tv_nsec;
  }
  return temp;
}

int main(int argc, char *argv[]) {
  if (argc < 3) {
    printf("error:Need more argument\n");
    return 0;
  }
  int gridsize = atoi(argv[1]);
  int blocksize = atoi(argv[2]);
  int N;
  Node *sl;
  Node *d_sl;
  Node *n_arr;
  Node *d_n_arr;
  struct timespec time1, time2, temp;
  int loop;
  // initializtion
  double time_used,sum=0;
  N = gridsize * blocksize / MAX_LEVEL;
  for(loop=1;loop<=1;loop++){
  int *input = (int *)malloc(N * sizeof(int));
  for (int i = 0; i < N; i++) {
    input[i] = i;
  }
  srand(time(NULL));
  shuffle(input,N);

  thrust::host_vector<int> h_s(N);
    //give number to host_vector
    for(int i=0;i<N;i++){
    	h_s[i]=input[i];
    }
    thrust::device_vector<int> d_s(h_s);

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
    thrust::sort(d_s.begin(), d_s.end()); // sorting
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&time2);
    temp = diff(time1, time2);
    time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
    printf("Sorting time= %f\n", time_used);
    //copy device to host
    h_s=d_s;
    //check
    for(int i=0;i<N;i++)
	    assert(h_s[i]==i);
  sl = (Node *)malloc(N * MAX_LEVEL * sizeof(Node));
  n_arr = (Node *)malloc(N * sizeof(Node));
  hipMalloc(&d_sl, N * MAX_LEVEL * sizeof(Node));
  hipMalloc(&d_n_arr, N * sizeof(Node));
  for (int i = 0; i < MAX_LEVEL * N; i++) {
    sl[i].key = -1;
    sl[i].level = 0;
    sl[i].nextIdx = -1;
  }

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    n_arr[i].key = h_s[i];
    n_arr[i].level = rand() % MAX_LEVEL + 1;
  }

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1); // timespec start
  hipMemcpy(d_sl, sl, N * MAX_LEVEL * sizeof(Node), hipMemcpyHostToDevice);
  hipMemcpy(d_n_arr, n_arr, N * sizeof(Node), hipMemcpyHostToDevice);
  test_Init(d_sl, d_n_arr, N, gridsize, blocksize);
  test_Connect(d_sl, N, gridsize, blocksize);
  hipMemcpy(sl, d_sl, N * MAX_LEVEL * sizeof(Node), hipMemcpyDeviceToHost);
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2); // timespec stop

  /*printf("Skiplist node value:\n");
  for(int i=0 ; i<MAX_LEVEL*N ;i++){
          printf("%2d ",sl[i].key);
          if(i%N==N-1)
          printf("\n");
  }
  printf("Skiplist nextIdx:\n");
  for(int i=0 ;i<MAX_LEVEL*N;i++){
          printf("%2d ",sl[i].nextIdx%N);
          if(i%N==N-1)
                  printf("\n");
  }*/
  temp = diff(time1, time2);
  time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
  sum+=time_used;
  free(input);
  free(sl);
  free(n_arr);
  hipFree(d_sl);
  hipFree(d_n_arr);
  }
  printf("%d\t%f\n", N, sum/loop);
}
